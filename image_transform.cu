#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Magick++.h>
#include <iostream>
#include <string>
using namespace std; 
using namespace Magick;

#define BLOCK_SIZE 32
#define RANGE 256

typedef struct pixelRGB {
  int r;
  int g;
  int b;
} pixelRGB;

__global__ void image_to_grayscale(pixelRGB* pixels) {
  //Below equation found here: http://www.mathworks.com/matlabcentral/answers/99136-how-do-i-convert-my-rgb-image-to-grayscale-without-using-the-image-processing-toolbox?
  //intensity = 0.2989*red + 0.5870*green + 0.1140*blue
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  pixelRGB color = pixels[index];
  int intensity = (0.2989 * color.r) + (0.5870 * color.g) + (0.1140 * color.b);
  pixels[index].r = intensity;
  pixels[index].g = intensity;
  pixels[index].b = intensity;
  
}


int main (int argc, char** argv) {
  InitializeMagick(*argv);
  
  printf("In main\n");
  
  Image image;
  string filename ("bowtie.jpg");
  try {
    // Read a file into image object
    image.read(filename);
  }
  catch(Exception &error_) {
    cout << "Caught exception: " << error_.what() << endl;
    return 1;
  }
   int width = image.columns();
   int height = image.rows();
   printf("width: %d, height: %d\n", width, height);
   PixelPacket* cpu_packet = image.getPixels(0, 0, width, height);
   pixelRGB cpu_pixels[width*height];
   printf("Got pixels?\n");
   
   for (int i = 0; i < width; i++) {
     for(int j = 0; j < height; j++) {
       Color color = cpu_packet[j * width + i];
       cpu_pixels[j* width + i].r = color.redQuantum() / RANGE;
       cpu_pixels[j* width + i].g = color.greenQuantum() / RANGE;
       cpu_pixels[j* width + i].b = color.blueQuantum() / RANGE;
     }
   }
   
   
   // Color color = cpu_packet[0];
   // cout << (color.redQuantum() / range) << endl;
 
   pixelRGB* gpu_pixels;
   if(hipMalloc(&gpu_pixels, sizeof(pixelRGB) * width * height) != hipSuccess) {
    fprintf(stderr, "Failed to create image for the gpu\n");
    exit(2);
  }
  
  //Copy contents from cpu to gpu
   if(hipMemcpy(gpu_pixels, cpu_pixels, sizeof(pixelRGB) *  width * height, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy image from CPU to the GPU\n");
      }

   printf("Gottem\n");
   
   int blocks = (width * height + BLOCK_SIZE - 1) / BLOCK_SIZE;
   image_to_grayscale<<<blocks, BLOCK_SIZE>>>(gpu_pixels);

   hipError_t err = hipDeviceSynchronize();
   if(err != hipSuccess) {
     printf("\n%s\n", hipGetErrorString(err));
     fprintf(stderr, "\nFailed to synchronize correctly\n");
   }

   if(hipMemcpy(cpu_pixels, gpu_pixels, sizeof(pixelRGB) * width * height, hipMemcpyDeviceToHost) != hipSuccess) {
     fprintf(stderr, "Failed to copy gpu pixels to host\n");
   }

    printf("they've returned\n");
   
   for (int i = 0; i < width; i++) {
     for(int j = 0; j < height; j++) {
       pixelRGB temp = cpu_pixels[j* width + i];
       cpu_packet[j * width + i] = Color(temp.r, temp.g, temp.b);
     }
   }

   image.syncPixels();

   image.write("grayscale_" + filename);
   
  return 0;
}
