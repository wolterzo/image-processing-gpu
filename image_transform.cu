#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <Magick++.h>
#include <iostream>
#include <string>
using namespace std; 
using namespace Magick;

#define BLOCK_SIZE 32
#define RANGE 256

typedef struct pixelRGB {
  int r;
  int g;
  int b;
} pixelRGB;

__global__ void image_to_grayscale(pixelRGB* pixels) {
  //Below equation found here: http://www.mathworks.com/matlabcentral/answers/99136-how-do-i-convert-my-rgb-image-to-grayscale-without-using-the-image-processing-toolbox?
  //intensity = 0.2989*red + 0.5870*green + 0.1140*blue
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  pixelRGB color = pixels[index];
  double intensity = ((double) 0.2989 * color.r) + ((double) 0.5870 * color.g) + ((double) 0.1140 * color.b);
  //printf("%lf\n", intensity);
  pixels[index].r = intensity;
  pixels[index].g = intensity;
  pixels[index].b = intensity;
  //printf("%d\n", pixels[index].r);
}

__global__ void matrix_filter_image(pixelRGB* input_pixels, pixelRGB* output_pixels, int w, int h, double* filter, int fWidth, int fHeight, double factor, double bias) {
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  
  int x = index % w;
  int y = index / w;

  
  // Looked at 'http://lodev.org/cgtutor/filtering.html' to figure out complexfilters

  double red = 0;
  double green = 0;
  double blue = 0;

  for(int fY = 0; fY < fHeight; fY++) {
    for(int fX = 0; fX < fWidth; fX++) {
      int imageX = (x - fWidth / 2 + fX + w) % w;
      int imageY = (y - fHeight / 2 + fY + h) % h;
      red += input_pixels[imageY * w + imageX].r * filter[fY * fWidth+fX];
      green += input_pixels[imageY * w + imageX].g * filter[fY * fWidth+fX];
      blue += input_pixels[imageY * w + imageX].b * filter[fY * fWidth+fX];
    }
  }
  // printf("In loop\n");
  output_pixels[index].r = min(max(int(factor * red + bias), 0), 255 * RANGE);
  output_pixels[index].g = min(max(int(factor * green + bias), 0), 255 * RANGE);
  output_pixels[index].b = min(max(int(factor * blue + bias), 0), 255 * RANGE);
}

int main (int argc, char** argv) {
  InitializeMagick(*argv);
  
  printf("In main\n");
  
  string filename = argv[1];//("bridge.jpg");
  Image image(filename);

  //START TIMER
  clock_t start = clock();


  //Filter
  int fHeight = 9;
  int fWidth = 9;

  double cpu_filter[] =
    {
      1, 0, 0, 0, 0, 0, 0, 0, 0,
      0, 1, 0, 0, 0, 0, 0, 0, 0,
      0, 0, 1, 0, 0, 0, 0, 0, 0,
      0, 0, 0, 1, 0, 0, 0, 0, 0,
      0, 0, 0, 0, 1, 0, 0, 0, 0,
      0, 0, 0, 0, 0, 1, 0, 0, 0,
      0, 0, 0, 0, 0, 0, 1, 0, 0,
      0, 0, 0, 0, 0, 0, 0, 1, 0,
      0, 0, 0, 0, 0, 0, 0, 0, 1,
    };
    
    // {
    //   -1, -1,  0,
    //   -1,  0,  1,
    //    0,  1,  1
    // };
  double factor = 1.0 / 9.0;
  double bias = 0.0;

  double* gpu_filter;
  if(hipMalloc(&gpu_filter, sizeof(double) * fWidth* fHeight) != hipSuccess) {
    fprintf(stderr, "Failed to create filter matrix for the gpu\n");
    exit(2);
  }
  
  //Copy contents from cpu to gpu
  if(hipMemcpy(gpu_filter, cpu_filter, sizeof(double) * fWidth * fHeight, hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Failed to copy filter matrix from CPU to the GPU\n");
  }
  
  int width = image.columns();
  int height = image.rows();
  printf("width: %d, height: %d\n", width, height);
  // Rounds up number of iterations
  int offset = fHeight / 2;
  int iterations = (((width * (height + offset)) + (65535 * BLOCK_SIZE))-1) / (65535 * BLOCK_SIZE);
  printf("number of iterations: %d\n", iterations);
  int modheight = height / iterations;
  int startheight = height / iterations;
  int modheight_plus;
  int startheight_plus;
  printf("modheight is: %d\n", modheight);
  pixelRGB* master_pixels;
  master_pixels = (pixelRGB*) malloc(sizeof(pixelRGB) * width * height);
  
  
  for (int i = 0; i < iterations; i++){
    int iter = i;
    int remainder = height - (modheight * i);
    if (remainder < modheight && remainder != 0) {
      printf("i is %d\n", i);
      printf("rm is %d\n", remainder);
      modheight = remainder;

      printf("On last iteration!! modheight = %d\n", modheight);
    }

    if(iterations == 1) {
      startheight_plus = 0;
      modheight_plus = modheight;
    }
    else if(iter == 0) {
      startheight_plus = 0;
      modheight_plus = modheight + offset;
    } else if(iter == iterations - 1) {
      startheight_plus = iter*startheight - offset;
      modheight_plus = modheight + offset;
    }
    else {
      startheight_plus =  iter*startheight - offset;
      modheight_plus = modheight + 2 * offset;
      printf("middle loop\n");
    }
    
    image.modifyImage();
    
    PixelPacket* cpu_packet = image.getPixels(0, startheight_plus, width, modheight_plus);
    printf("start height: %d, end height: %d\n", startheight_plus, modheight_plus);
    pixelRGB* cpu_pixels;
    cpu_pixels = (pixelRGB*) malloc(sizeof(pixelRGB) * width * modheight_plus);
    printf("Got pixels?\n");
   
    for (int i = 0; i < width; i++) {
      for(int j = 0; j < modheight_plus; j++) {
        Color color = cpu_packet[j * width + i];
        cpu_pixels[j* width + i].r = color.redQuantum();// / RANGE;
        cpu_pixels[j* width + i].g = color.greenQuantum();// / RANGE;
        cpu_pixels[j* width + i].b = color.blueQuantum();// / RANGE;
      }
    }
   

 
    pixelRGB* gpu_pixels;
    if(hipMalloc(&gpu_pixels, sizeof(pixelRGB) * width * modheight_plus) != hipSuccess) {
      fprintf(stderr, "Failed to create image for the gpu\n");
      exit(2);
    }
  
    //Copy contents from cpu to gpu
    if(hipMemcpy(gpu_pixels, cpu_pixels, sizeof(pixelRGB) *  width * modheight_plus, hipMemcpyHostToDevice) != hipSuccess) {
      fprintf(stderr, "Failed to copy image from CPU to the GPU\n");
    }

    printf("Gottem\n");
   
    pixelRGB* result_pixels;
    if(hipMalloc(&result_pixels, sizeof(pixelRGB) * width * modheight_plus) != hipSuccess) {
      fprintf(stderr, "Failed to create image for the gpu\n");
      exit(2);
    }
  
    //Copy contents from cpu to gpu
    if(hipMemcpy(result_pixels, cpu_pixels, sizeof(pixelRGB) *  width * modheight_plus, hipMemcpyHostToDevice) != hipSuccess) {
      fprintf(stderr, "Failed to copy image from CPU to the GPU\n");
    }


    int blocks = (width * modheight_plus + BLOCK_SIZE - 1) / BLOCK_SIZE;
    matrix_filter_image<<<blocks, BLOCK_SIZE>>>(gpu_pixels, result_pixels, width, modheight, gpu_filter, fWidth, fHeight, factor, bias);
    
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess) {
      printf("\n%s\n", hipGetErrorString(err));
      fprintf(stderr, "\nFailed to synchronize correctly\n");
    }

    if(hipMemcpy(cpu_pixels, result_pixels, sizeof(pixelRGB) * width * modheight_plus, hipMemcpyDeviceToHost) != hipSuccess) {
      fprintf(stderr, "Failed to copy gpu pixels to host\n");
    }

    printf("they've returned\n");

    int row_start = 1;
    if(iter == 0) {
      row_start = 0;
    }
    
    for (int i = row_start; i < modheight; i++) {
      for(int j = 0; j < width; j++) {
        int index = (startheight_plus * width) + (i*width) + j;
        master_pixels[index] = cpu_pixels[i* width + j];
      }
    }
    

    //image.syncPixels();
    free(cpu_pixels);
    hipFree(gpu_pixels);
    hipFree(result_pixels);
  }

  PixelPacket* all_packets = image.getPixels(0, 0, width, height);
  for (int i = 0; i < width; i++) {
    for(int j = 0; j < height; j++) {
      pixelRGB temp = master_pixels[j* width + i];
      all_packets[j * width + i] = Color(temp.r, temp.g, temp.b);
    }
  }
  
  image.syncPixels();
  image.write("filtered_" + filename);

  clock_t diff = clock() - start;
  int msec = diff * 1000 / CLOCKS_PER_SEC;

  FILE* timing = fopen("timing.csv", "a");
  fprintf(timing, "%d,%d\n", width*height, msec);
  fclose(timing);
  
  // free(cpu_pixels);
  return 0;
}
